
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_id() {
  printf("(%d %d %d) (%d %d %d)\n", blockIdx.x, blockIdx.y, blockIdx.z,
         threadIdx.x, threadIdx.y, threadIdx.z);
}

int main(void) {
  // specify grid and block dimensions
  dim3 dimGrid(2, 2, 2);
  dim3 dimBlock(2, 2, 2);

  // launch kernel
  print_id<<<dimGrid, dimBlock>>>();

  // ensure the kernel is launched before program exits
  hipDeviceSynchronize();

  return 0;
}
