
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_id() {
  printf("(%d %d %d) (%d %d %d)\n", blockIdx.x, blockIdx.y, blockIdx.z,
         threadIdx.x, threadIdx.y, threadIdx.z);
}

int main(void) {
  int N = 1000000;
  int grid = N / 1024;
  // specify grid and block dimensions
  dim3 dimGrid(grid, 1, 1);
  dim3 dimBlock(1024, 1, 1);

  // launch kernel
  print_id<<<dimGrid, dimBlock>>>();

  // ensure the kernel is launched before program exits
  hipDeviceSynchronize();

  return 0;
}
