
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void add_one(int *a, int n) {
  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (thread_id < n) { // Why is this necessary?
    a[thread_id] += 1;
  }
}

int main() {
  int n = 50;

  // Allocate memory on the host and initialize it
  int *a = (int *)malloc(n * sizeof(int));
  for (int i = 0; i < n; i++) {
    a[i] = i;
  }

  // Allocate memory on the device
  int *d_a;
  hipMalloc(&d_a, n * sizeof(int));
  hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);

  // Launch kernel
  add_one<<<5, 10>>>(d_a, n);

  // Copy result back to host
  hipMemcpy(a, d_a, n * sizeof(int), hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_a);

  // Print result
  for (int i = 0; i < n; i++) {
    printf("%d ", a[i]);
  }
  printf("\n");

  // Free host memory
  free(a);

  return 0;
}
