
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_id() {
  printf("(%d %d %d) (%d %d %d)\n", blockIdx.x, blockIdx.y, blockIdx.z,
         threadIdx.x, threadIdx.y, threadIdx.z);
}

int main(void) {
  // Specify grid and block dimensions
  // These support 3D grids and blocks, but we'll only use 1D for now
  dim3 dimGrid(2, 4, 8);
  dim3 dimBlock(1, 2, 3);

  // launch kernel
  print_id<<<dimGrid, dimBlock>>>();

  // ensure the kernel is launched before program exits
  hipDeviceSynchronize();

  return 0;
}
