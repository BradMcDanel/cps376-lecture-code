
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void add(int *a, int *b, int *c, int n) {
  // TODO: add `a` and `b` and store the result in `c`
  //       use `n` to determine the size of the arrays (number of elements)
}

int main() {
  int n = 50;
  int *a = (int *)malloc(n * sizeof(int));
  int *b = (int *)malloc(n * sizeof(int));
  int *c = (int *)malloc(n * sizeof(int));

  // initialize `a`, `b`, and `c` with some values
  for (int i = 0; i < n; i++) {
    a[i] = i;
    b[i] = n - i;
  }

  // (1) allocate device memory for `a`, `b`, and `c`
  // (2) copy `a` and `b` to the device
  // (3) launch the kernel
  //     Hint: how many blocks and threads do you need?
  // (4) copy `c` back to the host
  // (5) print the result
  // (6) free device memory
  // (7) free host memory
}