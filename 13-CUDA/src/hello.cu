
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_cuda() {
  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  printf("Hello from %d\n", thread_id);
}

int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("Usage: %s <number of threads>\n", argv[0]);
    return 1;
  }

  int num_threads = atoi(argv[1]);

  hello_cuda<<<1, num_threads>>>();

  // ensure the kernel is launched before program exits
  hipDeviceSynchronize();

  return 0;
}
