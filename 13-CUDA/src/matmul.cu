
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void matmul(float *a, float *b, float *c, int m, int n, int k) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  // this could be much more efficient!
  if (i < m && j < n) {
    float sum = 0;
    for (int p = 0; p < k; p++) {
      sum += a[i * k + p] * b[p * n + j];
    }
    c[i * n + j] = sum;
  }
}

int main(void) {
  int m = 1024;
  int n = 1024;
  int k = 1024;

  float *a, *b, *c;
  float *d_a, *d_b, *d_c;

  // allocate host memory
  a = (float *)malloc(m * k * sizeof(float));
  b = (float *)malloc(k * n * sizeof(float));
  c = (float *)malloc(m * n * sizeof(float));

  // allocate device memory
  hipMalloc((void **)&d_a, m * k * sizeof(float));
  hipMalloc((void **)&d_b, k * n * sizeof(float));
  hipMalloc((void **)&d_c, m * n * sizeof(float));

  // initialize the matrices on the host
  for (int i = 0; i < m * k; i++) {
    a[i] = 1.0f;
  }

  for (int i = 0; i < k * n; i++) {
    b[i] = 1.0f;
  }

  // copy to device
  hipMemcpy(d_a, a, m * k * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, k * n * sizeof(float), hipMemcpyHostToDevice);

  dim3 dimGrid(m / 32, n / 32, 1);
  dim3 dimBlock(32, 32, 1);

  matmul<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n, k);

  // copy back to host
  hipMemcpy(c, d_c, m * n * sizeof(float), hipMemcpyDeviceToHost);

  // verify the result
  for (int i = 0; i < m * n; i++) {
    if ((c[i] - k) > 1e-8) { // since all elements are 1.0f, this should be true
      printf("error: c[%d] = %f\n", i, c[i]);
      return -1;
    }
  }

  return 0;
}
