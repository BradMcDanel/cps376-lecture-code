
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void add(int *a, int *b, int *c, int n) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < n) {
    c[index] = a[index] + b[index];
  }
}

int main() {
  int n = 50;
  int *a = (int *)malloc(n * sizeof(int));
  int *b = (int *)malloc(n * sizeof(int));
  int *c = (int *)malloc(n * sizeof(int));

  // initialize `a`, `b`, and `c` with some values
  for (int i = 0; i < n; i++) {
    a[i] = i;
    b[i] = n - i;
  }

  // (1) allocate device memory for `a`, `b`, and `c`
  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, n * sizeof(int));
  hipMalloc(&d_b, n * sizeof(int));
  hipMalloc(&d_c, n * sizeof(int));

  // (2) copy `a` and `b` to the device
  hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

  // (3) launch the kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
  add<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

  // (4) copy `c` back to the host
  hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

  // (5) print the result
  for (int i = 0; i < n; i++) {
    printf("%d + %d = %d\n", a[i], b[i], c[i]);
  }

  // (6) free device memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  // (7) free host memory
  free(a);
  free(b);
  free(c);

  return 0;
}
