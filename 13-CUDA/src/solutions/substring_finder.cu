// Parallel substring finder
// returns the index of the first occurrence of the substring in the string (per
// thread) or -1 if not found


#include <hip/hip_runtime.h>
#include <random>
#include <string>

#include <stdio.h>
#include <stdlib.h>

char *random_string(int length) {
  // Create a random number generator
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_int_distribution<> dis(0, 25);
  char *str = (char *)malloc(length + 1);
  for (int i = 0; i < length; i++) {
    str[i] = 'a' + dis(gen);
  }
  str[length] = '\0';
  return str;
}

void __global__ find_substring(char *string, char *substring, int *result,
                               int string_length, int substring_length,
                               int num_threads) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int start_index = i * num_threads;
  int end_index = start_index + num_threads;
  int found = 0;
  int index = -1;

  printf("Thread %d: start_index = %d, end_index = %d, string_len = %d\n", i,
         start_index, end_index, string_length);

  if (end_index > string_length) {
    end_index = string_length;
  }

  for (int j = start_index; j < end_index; j++) {
    if (string[j] == substring[0]) {
      found = 1;
      for (int k = 1; k < substring_length; k++) {
        if (string[j + k] != substring[k]) {
          found = 0;
          break;
        }
      }
      if (found) {
        index = j;
        break;
      }
    }
  }

  result[i] = index;
}

int main(int argc, char *argv[]) {
  int string_length = 10000000;
  int substring_length = 5;
  int num_threads = 4;
  char *string = random_string(string_length);
  char *substring = (char *)"pizza";

  int *result = (int *)malloc(sizeof(int) * num_threads);
  int *d_result;
  hipMalloc((void **)&d_result, sizeof(int) * num_threads);

  char *d_string;
  hipMalloc((void **)&d_string, sizeof(char) * string_length);
  hipMemcpy(d_string, string, sizeof(char) * string_length,
             hipMemcpyHostToDevice);

  char *d_substring;
  hipMalloc((void **)&d_substring, sizeof(char) * substring_length);
  hipMemcpy(d_substring, substring, sizeof(char) * substring_length,
             hipMemcpyHostToDevice);

  find_substring<<<1, num_threads>>>(d_string, d_substring, d_result,
                                     string_length, substring_length,
                                     string_length / num_threads);

  hipMemcpy(result, d_result, sizeof(int) * num_threads,
             hipMemcpyDeviceToHost);

  int i;
  int min_index = -1;
  for (i = 0; i < num_threads; i++) {
    if (result[i] != -1) {
      min_index = result[i];
      break;
    }
  }

  if (min_index != -1) {
    printf("Found substring at index %d\n", min_index);
    char *found = (char *)malloc(sizeof(char) * (substring_length + 1));
    for (int j = 0; j < substring_length; j++) {
      found[j] = string[min_index + j];
    }
    found[substring_length] = '\0';

    printf("Result: %s\n", found);
  } else {
    printf("Substring not found\n");
  }

  // get a null-terminated string at the index
  hipFree(d_string);
  hipFree(d_substring);
  hipFree(d_result);
  free(string);
  free(result);

  return 0;
}
