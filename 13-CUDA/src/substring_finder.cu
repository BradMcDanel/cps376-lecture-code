// Parallel substring finder
// returns the index of the first occurrence of the substring in the string (per
// thread) or -1 if not found


#include <hip/hip_runtime.h>
#include <random>
#include <string>

#include <stdio.h>
#include <stdlib.h>

char *random_string(int length) {
  // Create a random number generator
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_int_distribution<> dis(0, 25);
  char *str = (char *)malloc(length + 1);
  for (int i = 0; i < length; i++) {
    str[i] = 'a' + dis(gen);
  }
  str[length] = '\0';
  return str;
}

void __global__ find_substring(char *string, char *substring, int *result,
                               int string_length, int substring_length,
                               int num_threads) {
  // This problem is significantly harder than the element-wise addition.
  // Talk with your neighbors about how you might solve this problem and
  // what each parametere means. Some things to chat about:
  // - How do you know which thread is responsible for which part of the string?
  // - How do you know when to stop searching?
  // - How do you return the result?
  // - How do you know when all threads have finished searching?
}

int main(int argc, char *argv[]) {
  int string_length = 10000000;
  int substring_length = 5;
  int num_threads = 4;
  char *string = random_string(string_length);
  char *substring = (char *)"pizza";

  int *result = (int *)malloc(sizeof(int) * num_threads);
  int *d_result;
  hipMalloc((void **)&d_result, sizeof(int) * num_threads);

  char *d_string;
  hipMalloc((void **)&d_string, sizeof(char) * string_length);
  hipMemcpy(d_string, string, sizeof(char) * string_length,
             hipMemcpyHostToDevice);

  char *d_substring;
  hipMalloc((void **)&d_substring, sizeof(char) * substring_length);
  hipMemcpy(d_substring, substring, sizeof(char) * substring_length,
             hipMemcpyHostToDevice);

  find_substring<<<1, num_threads>>>(d_string, d_substring, d_result,
                                     string_length, substring_length,
                                     string_length / num_threads);

  hipMemcpy(result, d_result, sizeof(int) * num_threads,
             hipMemcpyDeviceToHost);

  int i;
  int min_index = -1;
  for (i = 0; i < num_threads; i++) {
    if (result[i] != -1) {
      min_index = result[i];
      break;
    }
  }

  if (min_index != -1) {
    printf("Found substring at index %d\n", min_index);
    char *found = (char *)malloc(sizeof(char) * (substring_length + 1));
    for (int j = 0; j < substring_length; j++) {
      found[j] = string[min_index + j];
    }
    found[substring_length] = '\0';

    printf("Result: %s\n", found);
  } else {
    printf("Substring not found\n");
  }

  // get a null-terminated string at the index
  hipFree(d_string);
  hipFree(d_substring);
  hipFree(d_result);
  free(string);
  free(result);

  return 0;
}
