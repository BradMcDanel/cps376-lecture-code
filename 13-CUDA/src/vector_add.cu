
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

__global__ void vector_add(int *a, int *b, int *c, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    c[idx] = a[idx] + b[idx];
  }
}

void cpu_vector_add(int *a, int *b, int *c, int N) {
  for (int i = 0; i < N; i++) {
    c[i] = a[i] + b[i];
  }
}

int main(int argc, char **argv) {
  if (argc < 2) {
    printf("Usage: %s <N>\n", argv[0]);
    return 1;
  }

  int N = atoi(argv[1]);
  int *a, *b, *c, *c_ref;
  int *d_a, *d_b, *d_c;
  int size = N * sizeof(int);

  // Set the CUDA device
  hipSetDevice(0);

  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // Setup input values
  a = (int *)malloc(size);
  b = (int *)malloc(size);
  c = (int *)malloc(size);
  c_ref = (int *)malloc(size);

  // fill a and b with random values
  for (int i = 0; i < N; i++) {
    a[i] = rand() % 10;
    b[i] = rand() % 10;
  }

  // Copy input from host to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  int block_size = 256;
  dim3 dimGrid(N / block_size, 1, 1);
  dim3 dimBlock(block_size, 1, 1);

  // start timer
  clock_t start = clock();

  // call the kernel
  vector_add<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, N);

  // stop timer
  clock_t end = clock();

  // copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  double time_spent_gpu = (double)(end - start) / CLOCKS_PER_SEC;

  // call the CPU version
  clock_t start_cpu = clock();
  cpu_vector_add(a, b, c_ref, N);
  clock_t end_cpu = clock();
  double time_spent_cpu = (double)(end_cpu - start_cpu) / CLOCKS_PER_SEC;

  // compare results
  for (int i = 0; i < N; i++) {
    if (c[i] != c_ref[i]) {
      printf("Error: c[%d] = %d, c_ref[%d] = %d\n", i, c[i], i, c_ref[i]);
      return 1;
    }
  }

  printf("GPU time: %f\n", time_spent_gpu);
  printf("CPU time: %f\n", time_spent_cpu);

  // Cleanup
  free(a);
  free(b);
  free(c);
  free(c_ref);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  // Exit program
  return 0;
}
