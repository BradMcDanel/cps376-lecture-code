
#include <hip/hip_runtime.h>
#include "stdio.h"
#include "time.h"

#define TILE_SIZE 16

void cpu_matmul(float *a, float *b, float *c, int n) {
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      float sum = 0;
      for (int k = 0; k < n; k++) {
        sum += a[i * n + k] * b[k * n + j];
      }
      c[i * n + j] = sum;
    }
  }
}

__global__ void matmul(float *a, float *b, float *c, int n) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  // TODO: implement matrix multiplication
  // We assume matricies are square
  float sum = 0;
  for (int k = 0; k < n; k++) {
    sum += a[row * n + k] * b[k * n + col];
  }
  c[row * n + col] = sum;
}

int main(int argc, char **argv) {
  int n;
  if (argc == 2) {
    n = atoi(argv[1]);
  } else {
    printf("Usage: %s <matrix size>\n", argv[0]);
    return 0;
  }

  int size = n * n * sizeof(float);
  float *a, *b, *c, *c_gpu;
  float *d_a, *d_b, *d_c;
  a = (float *)malloc(size);
  b = (float *)malloc(size);
  c = (float *)malloc(size);
  c_gpu = (float *)malloc(size);
  hipMalloc(&d_a, size);
  hipMalloc(&d_b, size);
  hipMalloc(&d_c, size);
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      a[i * n + j] = 1.0;
      b[i * n + j] = 1.0;
    }
  }
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // CPU matmul
  clock_t start = clock();
  cpu_matmul(a, b, c, n);
  clock_t end = clock();
  double time_spent = (1000 * (double)(end - start) / CLOCKS_PER_SEC);
  printf("CPU matmul time: %f ms\n", time_spent);

  // Simple GPU matmul
  dim3 dimBlock(TILE_SIZE, TILE_SIZE);
  dim3 dimGrid(n / dimBlock.x, n / dimBlock.y);
  start = clock();
  matmul<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n);
  hipDeviceSynchronize();
  end = clock();
  time_spent = (1000 * (double)(end - start) / CLOCKS_PER_SEC);
  printf("Time for gpu matrix multiplication: %f ms\n", time_spent);
  hipMemcpy(c_gpu, d_c, size, hipMemcpyDeviceToHost);

  // Check results
  for (int i = 0; i < n * n; i++) {
    if (c[i] != c_gpu[i]) {
      printf("Error: c[%d] = %f, c_gpu[%d] = %f\n", i, c[i], i, c_gpu[i]);
      break;
    }
  }

  // Cleanup
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  free(a);
  free(b);
  free(c);
  free(c_gpu);

  return 0;
}