
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_id_1dgrid_1dblock() {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  printf("(%d) (%d) : %d\n", blockIdx.x, threadIdx.x, idx);
}

__global__ void print_id_1dgrid_2dblock() {
  // Use the 2D block and thread indices to calculate a unique 1D index
  // Then, print the 2D block and thread indices, and the 1D index

  // Hint: blockDim.x is the number of threads in a block in the x dimension
  //       blockDim.y is the number of threads in a block in the y dimension
  int idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
            threadIdx.x;
  printf("(%d, %d) (%d, %d) : %d\n", blockIdx.x, blockIdx.y, threadIdx.x,
         threadIdx.y, idx);
}

__global__ void print_id_1dgrid_3dblock() {
  // TODO: Implement this function
  // Hint: blockDim.z is the number of threads in a block in the z dimension
  int idx = blockIdx.x * blockDim.x * blockDim.y * blockDim.z +
            threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x +
            threadIdx.x;
  printf("(%d, %d, %d) (%d, %d, %d) : %d\n", blockIdx.x, blockIdx.y, blockIdx.z,
         threadIdx.x, threadIdx.y, threadIdx.z, idx);
}

int main(void) {
  //    grid:  x, y, z
  dim3 dimGrid(2, 1, 1);

  //    block:  x, y, z
  dim3 dimBlock(4, 1, 1);

  printf("1D grid, 1D block:\n");
  print_id_1dgrid_1dblock<<<dimGrid, dimBlock>>>();
  hipDeviceSynchronize(); // Do this between kernel launches to ensure that
                           // the previous kernel has finished executing

  // TODO: implement print_id_1dgrid_2dblock and print_id_1dgrid_3dblock
  dim3 dimGrid2(2, 1, 1);
  dim3 dimBlock2(2, 2, 1);
  printf("1D grid, 2D block:\n");
  print_id_1dgrid_2dblock<<<dimGrid2, dimBlock2>>>();
  hipDeviceSynchronize();

  dim3 dimGrid3(2, 1, 1);
  dim3 dimBlock3(2, 2, 3);
  printf("1D grid, 3D block:\n");
  print_id_1dgrid_3dblock<<<dimGrid3, dimBlock3>>>();
  hipDeviceSynchronize();

  return 0;
}
