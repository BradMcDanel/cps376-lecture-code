
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void init_rand_kernel(hiprandState *state) {
 int idx = blockIdx.x * blockDim.x + threadIdx.x;
 hiprand_init(0, idx, 0, &state[idx]);
}

__global__ void use_rand_kernel(hiprandState *state) {
 int idx = blockIdx.x * blockDim.x + threadIdx.x;
 float rnd = hiprand_uniform(&state[idx]);
 printf("(%d), %f\n", idx, rnd);
}

hiprandState* init_rand(int n_threads, int n_blocks) {
  hiprandState *d_state;
  hipMalloc(&d_state, n_threads * n_blocks * sizeof(hiprandState));
  init_rand_kernel<<<n_blocks, n_threads>>>(d_state);
  return d_state;
}

int main(void) {
  int n_threads = 16;
  int n_blocks = 1;
  hiprandState* d_state = init_rand(n_threads, n_blocks);
  use_rand_kernel<<<n_blocks, n_threads>>>(d_state);
  hipDeviceSynchronize();

  return 0;
}
