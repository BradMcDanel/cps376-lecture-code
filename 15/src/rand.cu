#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "utils.h"

__global__ void init_rand_kernel(hiprandState *state) {
 int idx = blockIdx.x * blockDim.x + threadIdx.x;
 hiprand_init(0, idx, 0, &state[idx]);
}


hiprandState* init_rand(int n_threads, int n_blocks) {
  hiprandState *d_state;
  hipMalloc(&d_state, n_threads * n_blocks * sizeof(hiprandState));
  init_rand_kernel<<<n_blocks, n_threads>>>(d_state);
  return d_state;
}




int main(void) {
  hiprandState* d_state = init_rand(n_threads, n_blocks);

  int n_threads = 1024;
  int n_blocks = 1;


  return 0;
}
